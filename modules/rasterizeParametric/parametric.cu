#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.h.cu"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "HostDeviceInterface.h"

#include "math.cuh"

constexpr int MAX_PATCHES = 1'000'000;

namespace cg = cooperative_groups;

// constexpr float uniformTime = 0.0;
Uniforms uniforms;
Allocator* allocator;

constexpr float PI = 3.1415;
constexpr uint32_t BACKGROUND_COLOR = 0x00332211ull;

struct Model{
	int functionID;
	float3 position;
};

struct Patch{
	float s_min;
	float s_max;
	float t_min;
	float t_max;
	int modelID;
};

float4 operator*(const mat4& a, const float4& b){
	return make_float4(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b),
		dot(a.rows[3], b)
	);
}

mat4 operator*(const mat4& a, const mat4& b){
	
	mat4 result;

	result.rows[0].x = dot(a.rows[0], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[0].y = dot(a.rows[0], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[0].z = dot(a.rows[0], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[0].w = dot(a.rows[0], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[1].x = dot(a.rows[1], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[1].y = dot(a.rows[1], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[1].z = dot(a.rows[1], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[1].w = dot(a.rows[1], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[2].x = dot(a.rows[2], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[2].y = dot(a.rows[2], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[2].z = dot(a.rows[2], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[2].w = dot(a.rows[2], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[3].x = dot(a.rows[3], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[3].y = dot(a.rows[3], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[3].z = dot(a.rows[3], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[3].w = dot(a.rows[3], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	return result;
}

void drawPoint(float4 coord, uint64_t* framebuffer, uint32_t color, Uniforms& uniforms){

	int x = coord.x;
	int y = coord.y;

	if(x > 1 && x < uniforms.width  - 2.0f)
	if(y > 1 && y < uniforms.height - 2.0f){

		// SINGLE PIXEL
		uint32_t pixelID = x + int(uniforms.width) * y;
		uint64_t udepth = *((uint32_t*)&coord.w);
		uint64_t encoded = (udepth << 32) | color;

		atomicMin(&framebuffer[pixelID], encoded);
	}
}

void drawSprite(float4 coord, uint64_t* framebuffer, uint32_t color, Uniforms& uniforms){

	int x = coord.x;
	int y = coord.y;

	if(x > 1 && x < uniforms.width  - 2.0f)
	if(y > 1 && y < uniforms.height - 2.0f){

		// POINT SPRITE
		for(int ox : {-2, -1, 0, 1, 2})
		for(int oy : {-2, -1, 0, 1, 2}){
			uint32_t pixelID = (x + ox) + int(uniforms.width) * (y + oy);
			uint64_t udepth = *((uint32_t*)&coord.w);
			uint64_t encoded = (udepth << 32) | color;

			atomicMin(&framebuffer[pixelID], encoded);
		}
	}
}

auto toScreen = [&](float3 p, Uniforms& uniforms){
	float4 ndc = uniforms.transform * float4{p.x, p.y, p.z, 1.0f};

	ndc.x = ndc.x / ndc.w;
	ndc.y = ndc.y / ndc.w;
	ndc.z = ndc.z / ndc.w;

	ndc.x = (ndc.x * 0.5f + 0.5f) * uniforms.width;
	ndc.y = (ndc.y * 0.5f + 0.5f) * uniforms.height;

	return ndc;
};

auto toScreen_locked = [&](float3 p, Uniforms& uniforms){
	float4 ndc = uniforms.locked_transform * float4{p.x, p.y, p.z, 1.0f};

	ndc.x = ndc.x / ndc.w;
	ndc.y = ndc.y / ndc.w;
	ndc.z = ndc.z / ndc.w;

	ndc.x = (ndc.x * 0.5f + 0.5f) * uniforms.width;
	ndc.y = (ndc.y * 0.5f + 0.5f) * uniforms.height;

	return ndc;
};

// s, t in range 0 to 1!
float3 sampleSphere(float s, float t){

	float u = 2.0f * 3.14f * s;
	float v = 3.14f * t;
	
	float3 xyz = {
		cos(u) * sin(v),
		sin(u) * sin(v),
		cos(v)
	};

	return xyz;
};

// s, t in range 0 to 1!
float3 samplePlane(float s, float t){
	return float3{2.0f * s - 1.0f, 0.0f, 2.0f * t - 1.0f};
};

float3 sampleFunkyPlane(float s, float t){

	float scale = 10.0f;
	float height = 0.105f;

	float time = uniforms.time;
	// float time = 123.0;
	float su = s - 0.5f;
	float tu = t - 0.5f;
	// float su = 1.0;
	// float tu = 1.0;
	float d = (su * su + tu * tu);

	// NOTE: It's very important for perf to explicitly specify float literals (e.g. 2.0f)
	float z = height * sin(scale * s + time) * cos(scale * t + time) 
	          + cos(2.0f * time) * 10.0f * height * exp(-1000.0f * d);

	return float3{
		2.0f * (-s + 0.5f), 
		z, 
		2.0f * (-t + 0.5f)
	};
};

float3 sampleExtraFunkyPlane(float s, float t){

	float scale = 10.0f;
	float height = 0.105f;

	float time = uniforms.time;
	// float time = 123.0;
	float su = s - 0.5f;
	float tu = t - 0.5f;
	// float su = 1.0;
	// float tu = 1.0;
	float d = (su * su + tu * tu);

	// NOTE: It's very important for perf to explicitly specify float literals (e.g. 2.0f)
	float z = height * sin(scale * s + time) * cos(scale * t + time) 
	    + cos(2.0f * time) * 10.0f * height * exp(-1000.0f * d)
	    + 0.002f * sin(2.0f * PI * 300.0f * s) * cos(2.0f * PI * 300.0f * t);

	return float3{
		2.0f * (-s + 0.5f), 
		z, 
		2.0f * (-t + 0.5f)
	};
};

// sampleSinCos, samplePlane, sampleSphere;
// auto sample = sampleSinCos;

auto getSampler(int model){
	if(model == MODEL_FUNKY_PLANE){
		return sampleFunkyPlane;
	}else if(model == MODEL_EXTRA_FUNKY_PLANE){
		return sampleExtraFunkyPlane;
	}else if(model == MODEL_SPHERE){
		return sampleSphere;
	}else{
		return samplePlane;
	}
};

void generatePatches2(
	Model* models, uint32_t* numModels, 
	Patch* patches, uint32_t* numPatches, 
	int threshold, 
	Uniforms& uniforms, 
	Stats* stats
){

	auto grid = cg::this_grid();

	if(grid.thread_rank() < 30){
		stats->numPatches[grid.thread_rank()] = 0;
	}
	
	Patch* patches_tmp_0 = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	Patch* patches_tmp_1 = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	uint32_t* numPatches_tmp_0 = allocator->alloc<uint32_t*>(4);
	uint32_t* numPatches_tmp_1 = allocator->alloc<uint32_t*>(4);

	struct PatchData{
		Patch* patch;
		uint32_t* counter;
	};

	PatchData* pingpong = allocator->alloc<PatchData*>(2 * sizeof(PatchData));
	pingpong[0] = {patches_tmp_0, numPatches_tmp_0};
	pingpong[1] = {patches_tmp_1, numPatches_tmp_1};

	if(grid.thread_rank() == 0){
		*numPatches_tmp_0 = 0;
		*numPatches_tmp_1 = 0;
	}

	grid.sync();

	// Create initial set of patches
	constexpr int initialPatchGridSize = 8;

	for(int modelID = 0; modelID < *numModels; modelID++){
		if(grid.thread_rank() < initialPatchGridSize * initialPatchGridSize){

			int index = grid.thread_rank();
			int ix = index % initialPatchGridSize;
			int iy = index / initialPatchGridSize;

			float s_min = float(ix + 0) / float(initialPatchGridSize);
			float s_max = float(ix + 1) / float(initialPatchGridSize);
			float t_min = float(iy + 0) / float(initialPatchGridSize);
			float t_max = float(iy + 1) / float(initialPatchGridSize);

			Patch patch = {s_min, s_max, t_min, t_max, modelID};

			patches_tmp_0[modelID * initialPatchGridSize * initialPatchGridSize + index] = patch;
		}
	}

	*numPatches_tmp_0 = *numModels * initialPatchGridSize * initialPatchGridSize;

	grid.sync();

	int level = 0;

	// SUBDIVIDE LARGE PATCHES
	// - if too large, divide and store in target
	// - if not too large, store in <patches>
	// - too large as in pixel size
	auto subdivide = [&](Patch* source, uint32_t* sourceCounter, Patch* target, uint32_t* targetCounter){

		processRange(*sourceCounter, [&](int index){
			Patch patch = source[index];

			float s_c = (patch.s_min + patch.s_max) * 0.5f;
			float t_c = (patch.t_min + patch.t_max) * 0.5f;

			Model model = models[patch.modelID];
			auto sample = getSampler(model.functionID);

			float3 p_00 = sample(patch.s_min, patch.t_min) + model.position;
			float3 p_01 = sample(patch.s_min, patch.t_max) + model.position;
			float3 p_10 = sample(patch.s_max, patch.t_min) + model.position;
			float3 p_11 = sample(patch.s_max, patch.t_max) + model.position;
			float3 p_c = sample(s_c, t_c) + model.position;


			float3 nodeMin = {
				min(min(min(p_00.x, p_01.x), min(p_10.x, p_11.x)), p_c.x),
				min(min(min(p_00.y, p_01.y), min(p_10.y, p_11.y)), p_c.y),
				min(min(min(p_00.z, p_01.z), min(p_10.z, p_11.z)), p_c.z),
			};
			float3 nodeMax = {
				max(max(max(p_00.x, p_01.x), max(p_10.x, p_11.x)), p_c.x),
				max(max(max(p_00.y, p_01.y), max(p_10.y, p_11.y)), p_c.y),
				max(max(max(p_00.z, p_01.z), max(p_10.z, p_11.z)), p_c.z),
			};
			bool isIntersectingFrustum = intersectsFrustum(uniforms.locked_transform, nodeMin, nodeMax);

			if(!isIntersectingFrustum){
				return;
			}

			float4 ps_00 = toScreen_locked(p_00, uniforms);
			float4 ps_01 = toScreen_locked(p_01, uniforms);
			float4 ps_10 = toScreen_locked(p_10, uniforms);
			float4 ps_11 = toScreen_locked(p_11, uniforms);
			float4 ps_c = toScreen_locked(p_c, uniforms);

			float min_x = min(min(min(ps_00.x, ps_01.x), min(ps_10.x, ps_11.x)), ps_c.x);
			float min_y = min(min(min(ps_00.y, ps_01.y), min(ps_10.y, ps_11.y)), ps_c.y);
			float max_x = max(max(max(ps_00.x, ps_01.x), max(ps_10.x, ps_11.x)), ps_c.x);
			float max_y = max(max(max(ps_00.y, ps_01.y), max(ps_10.y, ps_11.y)), ps_c.y);

			float s_x = max_x - min_x;
			float s_y = max_y - min_y;
			float area = s_x * s_y;

			if(area > threshold * threshold){
				// too large, subdivide into 4 smaller patches

				uint32_t targetIndex = atomicAdd(targetCounter, 4);

				if(targetIndex >= MAX_PATCHES) return;

				float s_center = (patch.s_min + patch.s_max) / 2.0f;
				float t_center = (patch.t_min + patch.t_max) / 2.0f;

				Patch patch_00 = {patch.s_min, s_center, patch.t_min, t_center, patch.modelID};
				Patch patch_01 = {patch.s_min, s_center, t_center, patch.t_max, patch.modelID};
				Patch patch_10 = {s_center, patch.s_max, patch.t_min, t_center, patch.modelID};
				Patch patch_11 = {s_center, patch.s_max, t_center, patch.t_max, patch.modelID};

				target[targetIndex + 0] = patch_00;
				target[targetIndex + 1] = patch_01;
				target[targetIndex + 2] = patch_10;
				target[targetIndex + 3] = patch_11;

			}else{
				// small enough, add to list of patches

				// TODO: do backface culling here? 
				// If the patch faces away from the camera, ignore it. 

				// float3 t_01 = p_01 - p_00;
				// float3 t_10 = p_10 - p_00;
				// float3 N = normalize(cross(t_01, t_10));
				// float3 N_v = make_float3(uniforms.view * float4{N.x, N.y, N.z, 0.0});
				
				// float a = dot(N_v, float3{0.0, 0.0, 1.0});
				// if(a < 0.0) return;

				uint32_t targetIndex = atomicAdd(numPatches, 1);

				if(targetIndex >= MAX_PATCHES) return;

				patches[targetIndex] = patch;

				atomicAdd(&stats->numPatches[level], 1);
			}

		});
	};

	grid.sync();

	// DIVIDE IN PING-PONG FASHION
	for(int i = 0; i < 14; i++){

		grid.sync();

		int sourceIndex = (i + 0) % 2;
		int targetIndex = (i + 1) % 2;

		PatchData source = pingpong[sourceIndex];
		PatchData target = pingpong[targetIndex];

		*target.counter = 0;

		grid.sync();

		subdivide(source.patch, source.counter, target.patch, target.counter);

		grid.sync();

		*target.counter = min(*target.counter, MAX_PATCHES);
		*numPatches = min(*numPatches, MAX_PATCHES);

		level++;
	}

}

// Rasterize a patch by sampling a 32x32 grid.
// - We launch with workgroup-size 1024, i.e., 32x32 threads
// - Therefore we ca let each thread process one sample of the patch concurrently
// - However, workgroup threads (unlike warp threads) don't operate simultaneously.
//      - So in order to compute the normal, we compute samples and store the results in shared memory
//      - Then we sync the group, and then each thread loads adjacent samples to compute the normal
void rasterizePatches_32x32(
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches, 
	uint64_t* framebuffer, 
	Uniforms& uniforms,
	Patch* newPatches, uint32_t* numNewPatches, 
	bool createNewPatches
){

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uint32_t& processedPatches = *allocator->alloc<uint32_t*>(4);
	if(grid.thread_rank() == 0){
		processedPatches = 0;
	}

	__shared__ int sh_patchIndex;
	__shared__ float sh_samples[1024 * 3];
	__shared__ int sh_pixelPositions[1024];
	__shared__ bool sh_NeedsRefinement;

	block.sync();

	int loop_max = 10'000;
	for(int loop_i = 0; loop_i < loop_max; loop_i++){

		// grab the index of the next unprocessed patch
		block.sync();
		if(block.thread_rank() == 0){
			sh_patchIndex = atomicAdd(&processedPatches, 1);
			sh_NeedsRefinement = false;
		}
		block.sync();

		if(sh_patchIndex >= *numPatches) break;

		Patch patch = patches[sh_patchIndex];
		Model model = models[patch.modelID];

		auto sample = getSampler(model.functionID);

		float s_min = patch.s_min;
		float s_max = patch.s_max;
		float t_min = patch.t_min;
		float t_max = patch.t_max;

		int index_t = block.thread_rank();
		int index_tx = index_t % 32;
		int index_ty = index_t / 32;

		float uts = float(index_tx) / 32.0f;
		float vts = float(index_ty) / 32.0f;

		float s = (1.0f - uts) * s_min + uts * s_max;
		float t = (1.0f - vts) * t_min + vts * t_max;

		float3 p = sample(s, t) + model.position;

		block.sync();

		// Store samples in shared memory, so that other threads can access them
		sh_samples[3 * index_t + 0] = p.x;
		sh_samples[3 * index_t + 1] = p.y;
		sh_samples[3 * index_t + 2] = p.z;

		block.sync();

		int inx = index_t + (index_tx < 31 ?  1 :  -1);
		int iny = index_t + (index_ty < 31 ? 32 : -32);
		int inxy = index_t;
		if(index_tx < 31) inxy += 1;
		if(index_ty < 31) inxy += 32;

		// Lead adjacent samples (next-x and next-y) to compute normal
		float3 pnx = {sh_samples[3 * inx + 0], sh_samples[3 * inx + 1], sh_samples[3 * inx + 2]};
		float3 pny = {sh_samples[3 * iny + 0], sh_samples[3 * iny + 1], sh_samples[3 * iny + 2]};

		float3 tx = normalize(pnx - p);
		float3 ty = normalize(pny - p);
		float3 N = normalize(cross(ty, tx));

		float4 ps = toScreen(p, uniforms);

		// Compute pixel positions and store them in shared memory so that ajdacent threads can access them
		uint32_t pixelPos; 
		int16_t* pixelPos_u16 = (int16_t*)&pixelPos;
		pixelPos_u16[0] = int(ps.x);
		pixelPos_u16[1] = int(ps.y);
		sh_pixelPositions[index_t] = pixelPos;

		block.sync();

		// compute pixel distances to next samples in x, y, or both directions
		uint32_t pp_00 = sh_pixelPositions[index_t];
		uint32_t pp_10 = sh_pixelPositions[inx];
		uint32_t pp_01 = sh_pixelPositions[iny];
		int16_t* pp_00_u16 = (int16_t*)&pp_00;
		int16_t* pp_10_u16 = (int16_t*)&pp_10;
		int16_t* pp_01_u16 = (int16_t*)&pp_01;

		// the max distance
		int d_max_10 = max(abs(pp_10_u16[0] - pp_00_u16[0]), abs(pp_10_u16[1] - pp_00_u16[1]));
		int d_max_01 = max(abs(pp_01_u16[0] - pp_00_u16[0]), abs(pp_01_u16[1] - pp_00_u16[1]));
		int d_max = max(d_max_10, d_max_01);

		uint32_t color = 0;
		// uint32_t color = patch.dbg * 12345678;
		uint8_t* rgba = (uint8_t*)&color;
		rgba[0] = 200.0f * N.x;
		rgba[1] = 200.0f * N.y;
		rgba[2] = 200.0f * N.z;
		rgba[3] = 255;

		// mark samples where distances to next samples are >1px
		if(index_tx < 31 && index_ty < 31)
		if(d_max > 1){
			// color = 0x00ff00ff;
			sh_NeedsRefinement = true;
		}

		block.sync();

		// if(sh_NeedsRefinement){
		// 	color = 0x0000ffff;
		// }

		if(!createNewPatches){
			color = 0x000000ff;
		}

		// color = (patch.x + 1) * (patch.y + 13) * 1234567;

		// drawSprite(ps, framebuffer, color, uniforms);

		// if(N.x > 10.0)
		drawPoint(ps, framebuffer, color, uniforms);

		block.sync();

		// If pixel distances are too large, create new patches to draw
		if(createNewPatches)
		if(sh_NeedsRefinement && block.thread_rank() == 0){

			uint32_t newPatchIndex = atomicAdd(numNewPatches, 4);

			if(newPatchIndex >= MAX_PATCHES){
				atomicSub(numNewPatches, 4);
				continue;
			}

			// marked as volatile to reduce register pressure and allow larger workgroup size
			volatile float s_center = (patch.s_min + patch.s_max) * 0.5f;
			volatile float t_center = (patch.t_min + patch.t_max) * 0.5f;

			newPatches[newPatchIndex + 0] = {
				patch.s_min, s_center,
				patch.t_min, t_center, 
				patch.modelID
			};

			newPatches[newPatchIndex + 1] = {
				s_center, patch.s_max,
				patch.t_min, t_center,
				patch.modelID
			};

			newPatches[newPatchIndex + 2] = {
				patch.s_min, s_center,
				t_center, patch.t_max,
				patch.modelID
			};

			newPatches[newPatchIndex + 3] = {
				s_center, patch.s_max,
				t_center, patch.t_max,
				patch.modelID
			};
		}
	}
}

// Unlike the 32x32 method, this method draws the patch "line by line".
// - We launch with 128 threads, which sample atx =threadID / 128.0 and y using the loop counter i. 
// - No normals yet, but we could probably sample the first 2x128 samples and compute normals, 
//   and then with each iteration next row of 128 samples and compute normals using previous 128 samples. 
void rasterizePatches_runnin_thru(
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	uint64_t* framebuffer, 
	Uniforms& uniforms
){

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uint32_t& processedPatches = *allocator->alloc<uint32_t*>(4);
	if(grid.thread_rank() == 0){
		processedPatches = 0;
	}

	__shared__ int sh_patchIndex;
	__shared__ float sh_samples[1024 * 3];

	block.sync();

	int loop_max = 10'000;
	for(int loop_i = 0; loop_i < loop_max; loop_i++){

		// grab the index of the next unprocessed triangle
		block.sync();
		if(block.thread_rank() == 0){
			sh_patchIndex = atomicAdd(&processedPatches, 1);
		}
		block.sync();

		if(sh_patchIndex >= *numPatches) break;

		Patch patch = patches[sh_patchIndex];
		Model model = models[patch.modelID];
		auto sample = getSampler(uniforms.model);

		float s_min = patch.s_min;
		float s_max = patch.s_max;
		float t_min = patch.t_min;
		float t_max = patch.t_max;

		int index_t = block.thread_rank();
		float ut = float(index_t) / float(block.num_threads());

		float s = (1.0f - ut) * s_min + ut * s_max;
		float t = t_min;

		
		float steps = 64.0f;
		for(float i = 0.0f; i < steps; i = i + 1.0f){
			float vt = i / steps;
			float t = (1.0f - vt) * t_min + vt * t_max;

			float3 p = sample(s, t) + model.position;
			uint32_t color = 0x000000ff;
			float4 ps = toScreen(p, uniforms);


			color = 1234567.0f * (123.0f + patch.s_min * patch.t_min);

			drawPoint(ps, framebuffer, color, uniforms);

		}


		

		block.sync();
	}
}

extern "C" __global__
void kernel_generate_scene(
	const Uniforms _uniforms,
	unsigned int* buffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uniforms = _uniforms;

	if(grid.thread_rank() == 0){

		models[0] = {uniforms.model, float3{ 2.1, 0.0, -2.1}};
		models[1] = {uniforms.model, float3{ 0.0, 0.0, -2.1}};
		models[2] = {uniforms.model, float3{-2.1, 0.0, -2.1}};

		models[3] = {uniforms.model, float3{ 2.1, 0.0, 0.0}};
		models[4] = {uniforms.model, float3{ 0.0, 0.0, 0.0}};
		models[5] = {uniforms.model, float3{-2.1, 0.0, 0.0}};

		models[6] = {uniforms.model, float3{ 2.1, 0.0, 2.1}};
		models[7] = {uniforms.model, float3{ 0.0, 0.0, 2.1}};
		models[8] = {uniforms.model, float3{-2.1, 0.0, 2.1}};

		*numModels = 9;
	}

}

extern "C" __global__
void kernel_generate_patches(
	const Uniforms _uniforms,
	unsigned int* buffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uniforms = _uniforms;

	Allocator _allocator(buffer, 0);
	allocator = &_allocator;

	grid.sync();
	if(grid.thread_rank() == 0){
		*numPatches = 0;
	}
	grid.sync();

	int threshold = 32;
	if(uniforms.method == METHOD_32X32){
		threshold = 32;
	}else if(uniforms.method == METHOD_RUNNIN_THRU){
		threshold = 64;
	}

	generatePatches2(models, numModels, patches, numPatches, threshold, uniforms, stats);


}

// Compute a whole lot of samples to check how many we can compute in a given time
// - We don't write the results to screen because that takes time, but we need to 
//   act as if we do so that the compiler doesn't optimize sample generation away.
// - Simply do some if that virtually never evaluates to true, and draw only if it's true. 
extern "C" __global__
void kernel_sampleperf_test(
	const Uniforms _uniforms,
	unsigned int* buffer,
	uint64_t* framebuffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uniforms = _uniforms;

	Allocator _allocator(buffer, 0);
	allocator = &_allocator;

	uint64_t t_00 = nanotime();

	grid.sync();


	auto sampler = getSampler(uniforms.model);
	int gridSize = 10'000; // 100M
	int numPixels = int(uniforms.width * uniforms.height);

	processRange(gridSize * gridSize, [&](int index){

		uint32_t ix = index % gridSize;
		uint32_t iy = index / gridSize;

		float s = float(ix) / float(gridSize);
		float t = float(iy) / float(gridSize);

		float3 sample = sampler(s, t);

		// Some bogus <if> that pretents to do something but virtually never evaluates to true,
		// so that sampler(...) isn't optimized away.
		if(sample.x * sample.y == 123.0f){
			int pixelID = int(sample.x * sample.y * 1234.0f) % numPixels;
			framebuffer[10'000] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
		}

	});

	grid.sync();

	uint64_t t_20 = nanotime();

	// TODO: should do timings in host code with events.
	if(grid.thread_rank() == 0 && (stats->frameID % 100) == 0){
		stats->time_0 = float((t_20 - t_00) / 1000llu) / 1000.0f;
		stats->time_1 = 0.0;
	}

}

extern "C" __global__
void kernel_clear_framebuffer(
	const Uniforms _uniforms,
	unsigned int* buffer,
	uint64_t* framebuffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){

	auto grid = cg::this_grid();

	processRange(0, _uniforms.width * _uniforms.height, [&](int pixelIndex){
		// framebuffer[pixelIndex] = 0x7f800000'00332211ull;
		framebuffer[pixelIndex] = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
	});

}

extern "C" __global__
void kernel_framebuffer_to_OpenGL(
	const Uniforms _uniforms,
	unsigned int* buffer,
	uint64_t* framebuffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){

	auto grid = cg::this_grid();

	// transfer framebuffer to opengl texture
	processRange(0, _uniforms.width * _uniforms.height, [&](int pixelIndex){

		int x = pixelIndex % int(_uniforms.width);
		int y = pixelIndex / int(_uniforms.width);

		uint64_t encoded = framebuffer[pixelIndex];
		uint32_t color = encoded & 0xffffffffull;

		surf2Dwrite(color, gl_colorbuffer, x * 4, y);
	});

	if(grid.thread_rank() == 0){
		stats->frameID++;
	}

}

extern "C" __global__
void kernel_rasterize_patches_32x32(
	const Uniforms _uniforms,
	unsigned int* buffer,
	uint64_t* framebuffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	Allocator _allocator(buffer, 0);

	uniforms = _uniforms;
	allocator = &_allocator;

	Patch* newPatches = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	uint32_t* numNewPatches = allocator->alloc<uint32_t*>(4);

	grid.sync();

	if(grid.thread_rank() == 0){
		*numNewPatches = 0;
	}

	grid.sync();

	rasterizePatches_32x32(
		models, numModels,
		patches, numPatches, 
		framebuffer, 
		uniforms,
		newPatches, numNewPatches, true
	);
	grid.sync();

	if(uniforms.enableRefinement){
		// the earlier call to rasterizePatches checked for holes and created
		// a refined list of patches. render them now. 
		rasterizePatches_32x32(
			models, numModels,
			newPatches, numNewPatches, 
			framebuffer, 
			uniforms,
			patches, numPatches, false
		);
		grid.sync();
	}

	grid.sync();

	

}

extern "C" __global__
void kernel_rasterize_patches_runnin_thru(
	const Uniforms _uniforms,
	unsigned int* buffer,
	uint64_t* framebuffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	Allocator _allocator(buffer, 0);

	uniforms = _uniforms;
	allocator = &_allocator;

	grid.sync();

	uint64_t t_00 = nanotime();

	rasterizePatches_runnin_thru(models, numModels, patches, numPatches, framebuffer, uniforms);
	grid.sync();

	uint64_t t_20 = nanotime();

	if(grid.thread_rank() == 0 && (stats->frameID % 100) == 0){
		stats->time_1 = float((t_20 - t_00) / 1000llu) / 1000.0f;
	}

}

// just some debugging. and checking how many registers a simple kernel utilizes.
extern "C" __global__
void kernel_test(
	const Uniforms _uniforms,
	unsigned int* buffer,
	Model* models, uint32_t* numModels,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();


	uint32_t size = uniforms.width * uniforms.height;
	uint32_t totalThreadCount = blockDim.x * gridDim.x;
	int itemsPerThread = size / totalThreadCount + 1;

	for(int i = 0; i < itemsPerThread; i++){
		int block_offset  = itemsPerThread * blockIdx.x * blockDim.x;
		int thread_offset = itemsPerThread * threadIdx.x;
		int index = block_offset + thread_offset + i;

		if(index >= size){
			break;
		}

		int x = index % int(uniforms.width);
		int y = index / int(uniforms.width);

		uint32_t color = 0x00112233;

		surf2Dwrite(color, gl_colorbuffer, x * 4, y);
	}

}
