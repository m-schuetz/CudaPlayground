#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.h.cu"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "HostDeviceInterface.h"

#include "math.cuh"

constexpr int MAX_PATCHES = 1'000'000;

float4 operator*(const mat4& a, const float4& b){
	return make_float4(
		dot(a.rows[0], b),
		dot(a.rows[1], b),
		dot(a.rows[2], b),
		dot(a.rows[3], b)
	);
}

mat4 operator*(const mat4& a, const mat4& b){
	
	mat4 result;

	result.rows[0].x = dot(a.rows[0], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[0].y = dot(a.rows[0], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[0].z = dot(a.rows[0], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[0].w = dot(a.rows[0], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[1].x = dot(a.rows[1], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[1].y = dot(a.rows[1], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[1].z = dot(a.rows[1], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[1].w = dot(a.rows[1], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[2].x = dot(a.rows[2], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[2].y = dot(a.rows[2], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[2].z = dot(a.rows[2], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[2].w = dot(a.rows[2], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	result.rows[3].x = dot(a.rows[3], {b.rows[0].x, b.rows[1].x, b.rows[2].x, b.rows[3].x});
	result.rows[3].y = dot(a.rows[3], {b.rows[0].y, b.rows[1].y, b.rows[2].y, b.rows[3].y});
	result.rows[3].z = dot(a.rows[3], {b.rows[0].z, b.rows[1].z, b.rows[2].z, b.rows[3].z});
	result.rows[3].w = dot(a.rows[3], {b.rows[0].w, b.rows[1].w, b.rows[2].w, b.rows[3].w});

	return result;
}

namespace cg = cooperative_groups;

Uniforms uniforms;
Allocator* allocator;
uint64_t nanotime_start;

constexpr float PI = 3.1415;
constexpr uint32_t BACKGROUND_COLOR = 0x00332211ull;

auto toScreen = [&](float3 p, Uniforms& uniforms){
	float4 ndc = uniforms.transform * float4{p.x, p.y, p.z, 1.0};

	ndc.x = ndc.x / ndc.w;
	ndc.y = ndc.y / ndc.w;
	ndc.z = ndc.z / ndc.w;

	ndc.x = (ndc.x * 0.5 + 0.5) * uniforms.width;
	ndc.y = (ndc.y * 0.5 + 0.5) * uniforms.height;
	// ndc.z = (ndc.z * 0.5 + 0.5) * uniforms.width;

	return ndc;
};

struct Patch{
	float s_min;
	float s_max;
	float t_min;
	float t_max;
};

void generatePatches2(Patch* patches, uint32_t* numPatches, Uniforms& uniforms){

	auto grid = cg::this_grid();

	
	Patch* patches_finished = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	Patch* patches_tmp_0 = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	Patch* patches_tmp_1 = allocator->alloc<Patch*>(MAX_PATCHES * sizeof(Patch));
	uint32_t* numPatches_finished = allocator->alloc<uint32_t*>(4);
	uint32_t* numPatches_tmp_0 = allocator->alloc<uint32_t*>(4);
	uint32_t* numPatches_tmp_1 = allocator->alloc<uint32_t*>(4);

	struct PatchData{
		Patch* patch;
		uint32_t* counter;
	};

	PatchData* pingpong = allocator->alloc<PatchData*>(2 * sizeof(PatchData));
	pingpong[0].patch = patches_tmp_0;
	pingpong[0].counter = numPatches_tmp_0;
	pingpong[1].patch = patches_tmp_1;
	pingpong[1].counter = numPatches_tmp_1;

	if(grid.thread_rank() == 0){
		*numPatches_finished = 0;
		*numPatches_tmp_0 = 0;
		*numPatches_tmp_1 = 0;
	}

	grid.sync();

	// if(grid.thread_rank() == 0) 
	// 	printf("target.counter: %llu \n", pingpong[2].counter);

	Patch root;
	root.s_min = 0;
	root.s_max = 1;
	root.t_min = 0;
	root.t_max = 1;

	patches_tmp_0[0] = root;
	*numPatches_tmp_0 = 1;

	// SUBDIVIDE LARGE PATCHES
	// - if too large, divide and store in target
	// - if not too large, store in <patches>
	auto subdivide = [&](Patch* source, uint32_t* sourceCounter, Patch* target, uint32_t* targetCounter){

		processRange(*sourceCounter, [&](int index){
			Patch patch = source[index];

			float3 p_00 = sample(patch.s_min, patch.t_min);
			float3 p_01 = sample(patch.s_min, patch.t_max);
			float3 p_10 = sample(patch.s_max, patch.t_min);
			float3 p_11 = sample(patch.s_max, patch.t_max);

			float3 t_01 = p_01 - p_00;
			float3 t_10 = p_10 - p_00;
			float3 N = normalize(cross(t_01, t_10));

			float4 V = uniforms.view * float4(0.0, 0.0, 1.0, 0.0);


			float3 nodeMin = {
				min(min(p_00.x, p_01.x), min(p_10.x, p_11.x)),
				min(min(p_00.y, p_01.y), min(p_10.y, p_11.y)),
				min(min(p_00.z, p_01.z), min(p_10.z, p_11.z)),
			};
			float3 nodeMax = {
				max(max(p_00.x, p_01.x), max(p_10.x, p_11.x)),
				max(max(p_00.y, p_01.y), max(p_10.y, p_11.y)),
				max(max(p_00.z, p_01.z), max(p_10.z, p_11.z)),
			};
			bool isIntersectingFrustum = intersectsFrustum(uniforms.transform, nodeMin, nodeMax);

			if(!isIntersectingFrustum){
				return;
			}

			float4 ps_00 = toScreen(p_00, uniforms);
			float4 ps_01 = toScreen(p_01, uniforms);
			float4 ps_10 = toScreen(p_10, uniforms);
			float4 ps_11 = toScreen(p_11, uniforms);

			float min_x = min(min(ps_00.x, ps_01.x), min(ps_10.x, ps_11.x));
			float max_x = max(max(ps_00.x, ps_01.x), max(ps_10.x, ps_11.x));
			float min_y = min(min(ps_00.y, ps_01.y), min(ps_10.y, ps_11.y));
			float max_y = max(max(ps_00.y, ps_01.y), max(ps_10.y, ps_11.y));

			float s_x = max_x - min_x;
			float s_y = max_y - min_y;
			float area = s_x * s_y;

			if(area > 64 * 64){
				// too large, subdivide
				uint32_t targetIndex = atomicAdd(targetCounter, 4);

				float s_center = (patch.s_min + patch.s_max) / 2.0;
				float t_center = (patch.t_min + patch.t_max) / 2.0;

				Patch patch_00;
				patch_00.s_min = patch.s_min;
				patch_00.s_max = s_center;
				patch_00.t_min = patch.t_min;
				patch_00.t_max = t_center;
				target[targetIndex + 0] = patch_00;

				Patch patch_01;
				patch_01.s_min = patch.s_min;
				patch_01.s_max = s_center;
				patch_01.t_min = t_center;
				patch_01.t_max = patch.t_max;
				target[targetIndex + 1] = patch_01;

				Patch patch_10;
				patch_10.s_min = s_center;
				patch_10.s_max = patch.s_max;
				patch_10.t_min = patch.t_min;
				patch_10.t_max = t_center;
				target[targetIndex + 2] = patch_10;

				Patch patch_11;
				patch_11.s_min = s_center;
				patch_11.s_max = patch.s_max;
				patch_11.t_min = t_center;
				patch_11.t_max = patch.t_max;
				target[targetIndex + 3] = patch_11;


			}else{
				// small enough, add to final list
				uint32_t targetIndex = atomicAdd(numPatches, 4);
				patches[targetIndex] = patch;
			}

		});
	};

	grid.sync();

	// DIVIDE IN PING-PONG FASHION
	for(int i = 0; i < 14; i++){

		grid.sync();

		int sourceIndex = (i + 0) % 2;
		int targetIndex = (i + 1) % 2;

		PatchData source = pingpong[sourceIndex];
		PatchData target = pingpong[targetIndex];

		*target.counter = 0;

		grid.sync();

		subdivide(source.patch, source.counter, target.patch, target.counter);

		grid.sync();

	}

}

void generatePatches(Patch* patches, uint32_t* numPatches, Uniforms& uniforms){

	int gridsize = 64;

	processRange(0, gridsize * gridsize, [&](int index){

		int patch_x = index % gridsize;
		int patch_y = index / gridsize;

		int ux_0 = patch_x + 0;
		int ux_1 = patch_x + 1;
		int uy_0 = patch_y + 0;
		int uy_1 = patch_y + 1;

		float u_0 = float(ux_0) / float(gridsize);
		float u_1 = float(ux_1) / float(gridsize);
		float v_0 = float(uy_0) / float(gridsize);
		float v_1 = float(uy_1) / float(gridsize);

		float3 p_00 = sample(u_0, v_0);
		float3 p_01 = sample(u_0, v_1);
		float3 p_10 = sample(u_1, v_0);
		float3 p_11 = sample(u_1, v_1);

		float3 nodeMin = {
			min(min(p_00.x, p_01.x), min(p_10.x, p_11.x)),
			min(min(p_00.y, p_01.y), min(p_10.y, p_11.y)),
			min(min(p_00.z, p_01.z), min(p_10.z, p_11.z)),
		};
		float3 nodeMax = {
			max(max(p_00.x, p_01.x), max(p_10.x, p_11.x)),
			max(max(p_00.y, p_01.y), max(p_10.y, p_11.y)),
			max(max(p_00.z, p_01.z), max(p_10.z, p_11.z)),
		};
		bool isIntersectingFrustum = intersectsFrustum(uniforms.transform, nodeMin, nodeMax);

		if(!isIntersectingFrustum){
			return;
		}

		float4 ps_00 = toScreen(p_00, uniforms);
		float4 ps_01 = toScreen(p_01, uniforms);
		float4 ps_10 = toScreen(p_10, uniforms);
		float4 ps_11 = toScreen(p_11, uniforms);

		float min_x = min(min(ps_00.x, ps_01.x), min(ps_10.x, ps_11.x));
		float max_x = max(max(ps_00.x, ps_01.x), max(ps_10.x, ps_11.x));
		float min_y = min(min(ps_00.y, ps_01.y), min(ps_10.y, ps_11.y));
		float max_y = max(max(ps_00.y, ps_01.y), max(ps_10.y, ps_11.y));

		float s_x = max_x - min_x;
		float s_y = max_y - min_y;
		float area = s_x * s_y;

		float4 p = ps_00;
		int x = p.x;
		int y = p.y;
		float depth = p.w;

		uint32_t R = 255.0f * u_0 / (2.0 * 3.14);
		uint32_t G = 255.0f * v_0 / 3.14;
		uint32_t B = 0;

		if(area < 32 * 32){
			R = 0;
			G = 255;
			B = 0;

			Patch patch;
			patch.s_min = u_0;
			patch.s_max = u_1;
			patch.t_min = v_0;
			patch.t_max = v_1;
			// patch.gridsize = gridsize;
			// patch.x = patch_x;
			// patch.y = patch_y;
			// patch.dbg = 0;

			uint32_t patchIndex = atomicAdd(numPatches, 1);
			patches[patchIndex] = patch;

		}else if(area < 64 * 64){
			R = 0;
			G = 0;
			B = 255;

			uint32_t patchIndex = atomicAdd(numPatches, 4);
			
			int i = 0; 
			for(int px : {0, 1})
			for(int py : {0, 1})
			{
				Patch patch;
				patch.s_min = float(2 * patch_x + px + 0) / float(2 * gridsize);
				patch.s_max = float(2 * patch_x + px + 1) / float(2 * gridsize);
				patch.t_min = float(2 * patch_y + py + 0) / float(2 * gridsize);
				patch.t_max = float(2 * patch_y + py + 1) / float(2 * gridsize);

				patches[patchIndex + i] = patch;

				i++;
			}
		}else if(area < 128 * 128){
			R = 255;
			G = 255;
			B = 0;

			uint32_t patchIndex = atomicAdd(numPatches, 16);
			
			int i = 0; 
			for(int px : {0, 1, 2, 3})
			for(int py : {0, 1, 2, 3})
			{
				Patch patch;
				// patch.gridsize = 4 * gridsize;
				// patch.x = 4 * patch_x + px;
				// patch.y = 4 * patch_y + py;
				// patch.dbg = 2;
				patch.s_min = float(4 * patch_x + px + 0) / float(4 * gridsize);
				patch.s_max = float(4 * patch_x + px + 1) / float(4 * gridsize);
				patch.t_min = float(4 * patch_y + py + 0) / float(4 * gridsize);
				patch.t_max = float(4 * patch_y + py + 1) / float(4 * gridsize);

				patches[patchIndex + i] = patch;

				i++;
			}
		}else if(area < 256 * 256){
			R = 255;
			G = 255;
			B = 255;

			uint32_t patchIndex = atomicAdd(numPatches, 64);
			
			int i = 0; 
			for(int px : {0, 1, 2, 3, 4, 5, 6, 7})
			for(int py : {0, 1, 2, 3, 4, 5, 6, 7})
			{
				Patch patch;
				patch.s_min = float(8 * patch_x + px + 0) / float(8 * gridsize);
				patch.s_max = float(8 * patch_x + px + 1) / float(8 * gridsize);
				patch.t_min = float(8 * patch_y + py + 0) / float(8 * gridsize);
				patch.t_max = float(8 * patch_y + py + 1) / float(8 * gridsize);

				patches[patchIndex + i] = patch;

				i++;
			}
		}else if(area < 512 * 512){
			R = 255;
			G = 0;
			B = 0;

			uint32_t patchIndex = atomicAdd(numPatches, 4 * 64);
			
			int i = 0; 
			for(int px : {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15})
			for(int py : {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15})
			{
				Patch patch;
				patch.s_min = float(16 * patch_x + px + 0) / float(16 * gridsize);
				patch.s_max = float(16 * patch_x + px + 1) / float(16 * gridsize);
				patch.t_min = float(16 * patch_y + py + 0) / float(16 * gridsize);
				patch.t_max = float(16 * patch_y + py + 1) / float(16 * gridsize);

				patches[patchIndex + i] = patch;

				i++;
			}
		}else{
			
			R = 255;
			G = 0;
			B = 0;

			uint32_t patchIndex = atomicAdd(numPatches, 16 * 64);
			
			int i = 0; 
			for(int px = 0; px < 32; px++)
			for(int py = 0; py < 32; py++)
			{
				Patch patch;
				patch.s_min = float(32 * patch_x + px + 0) / float(32 * gridsize);
				patch.s_max = float(32 * patch_x + px + 1) / float(32 * gridsize);
				patch.t_min = float(32 * patch_y + py + 0) / float(32 * gridsize);
				patch.t_max = float(32 * patch_y + py + 1) / float(32 * gridsize);

				patches[patchIndex + i] = patch;

				i++;
			}
		}
	});
}

extern "C" __global__
void kernel(
	const Uniforms _uniforms,
	unsigned int* buffer,
	Patch* patches, uint32_t* numPatches,
	hipSurfaceObject_t gl_colorbuffer,
	Stats* stats
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(nanotime_start));

	uniforms = _uniforms;

	Allocator _allocator(buffer, 0);
	allocator = &_allocator;

	grid.sync();
	if(grid.thread_rank() == 0){
		*numPatches = 0;
	}
	grid.sync();

	// generatePatches(patches, numPatches, uniforms);
	generatePatches2(patches, numPatches, uniforms);
}
