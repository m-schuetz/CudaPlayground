#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.cuh"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "HostDeviceInterface.h"

#include "triangles.cuh"
#include "lines.cuh"
#include "points.cuh"
#include "texture.cuh"
#include "voxels.cuh"
#include "globals.cuh"
#include "skybox.cuh"

namespace cg = cooperative_groups;


constexpr bool EDL_ENABLED = false;
constexpr uint32_t gridSize = 128;
constexpr float fGridSize = gridSize;
constexpr uint32_t numCells = gridSize * gridSize * gridSize;
constexpr float3 gridMin = { -1.0f, -1.0f, 0.0f};
constexpr float3 gridMax = { 1.0f, 1.0f, 2.0f};
constexpr float PI = 3.1415;
constexpr uint32_t BACKGROUND_COLOR = 0x00332211ull;

// struct Particle{
// 	float3 pos;
// 	uint32_t color;
// 	float3 velocity;
// };

constexpr int MAX_PARTICLES = 10'000'000;
// Particle particles[MAX_PARTICLES];

// struct{
// 	float3   position[MAX_PARTICLES];
// 	uint32_t color[MAX_PARTICLES];
// 	float    age[MAX_PARTICLES];
// 	float3   velocity[MAX_PARTICLES];
// } g_particles;


uint32_t SPECTRAL[11] = {
	0x42019e,
	0x4f3ed5,
	0x436df4,
	0x61aefd,
	0x8be0fe,
	0xbfffff,
	0x98f5e6,
	0xa4ddab,
	0xa5c266,
	0xbd8832,
	0xa24f5e,
};

extern "C" __global__
void kernel(
	const Uniforms _uniforms,
	uint32_t* buffer,
	hipSurfaceObject_t gl_colorbuffer_main,
	hipSurfaceObject_t gl_colorbuffer_vr_left,
	hipSurfaceObject_t gl_colorbuffer_vr_right,
	uint64_t* framebuffer,
	uint64_t* fb_vr_left,
	uint64_t* fb_vr_right,
	uint32_t numTriangles,
	float3* positions,
	float2* uvs,
	uint32_t* colors,
	uint32_t* textureData,
	const Skybox skybox
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(nanotime_start));

	uniforms = _uniforms;

	Allocator _allocator(buffer, 0);
	allocator = &_allocator;

	hiprandStateXORWOW_t thread_random_state;
	hiprand_init(grid.thread_rank(), 0, 0, &thread_random_state);

	{ // clear framebuffer
		uint64_t clearValue = (uint64_t(Infinity) << 32ull) | uint64_t(BACKGROUND_COLOR);
		clearBuffer_u64(framebuffer, clearValue, uniforms.width * uniforms.height);

		if(uniforms.vrEnabled){
			clearBuffer_u64(fb_vr_left, clearValue, uniforms.vr_left_width * uniforms.vr_left_height);
			clearBuffer_u64(fb_vr_right, clearValue, uniforms.vr_right_width * uniforms.vr_right_height);
		}
	}

	uint64_t* fb_points = allocator->alloc<uint64_t*>(uniforms.width * uniforms.height * sizeof(uint64_t));
	{
		uint64_t clearValue = (uint64_t(Infinity) << 32ull) | uint64_t(0);
		clearBuffer_u64(fb_points, clearValue, uniforms.width * uniforms.height);
	}

	struct{
		float3*   position;
		uint32_t* color;
		float*    age;
		float*    lifetime;
		float3*   velocity;
	} particles;

	particles.position = allocator->alloc<float3*  >(MAX_PARTICLES * sizeof(float3));
	particles.color    = allocator->alloc<uint32_t*>(MAX_PARTICLES * sizeof(uint32_t));
	particles.age      = allocator->alloc<float*   >(MAX_PARTICLES * sizeof(float));
	particles.lifetime = allocator->alloc<float*   >(MAX_PARTICLES * sizeof(float));
	particles.velocity = allocator->alloc<float3*  >(MAX_PARTICLES * sizeof(float3));



	// clear particles
	processRange(MAX_PARTICLES, [&](int index){
		uint32_t X = hiprand(&thread_random_state) >> 16;
		uint32_t Y = hiprand(&thread_random_state) >> 16;
		uint32_t Z = hiprand(&thread_random_state) >> 16;
		uint32_t upper = 1 << 16;

		uint32_t color = hiprand(&thread_random_state);

		color = SPECTRAL[color % 11];

		float x = 10.0f * (float(X) / float(upper) - 0.5f);
		float y =  1.0f * (float(Y) / float(upper) - 0.5f) - 0.8f;
		float z = 10.0f * (float(Z) / float(upper) - 0.5f);

		uint32_t lifetime_ms = hiprand(&thread_random_state) % 2000 + 1000;

		particles.position[index] = float3{x, y, z};
		particles.color[index] = color;
		particles.age[index] = 0.0f;
		particles.lifetime[index] = float(lifetime_ms) / 1000.0f;

		// g_particles.position[index] = float3{0.0f, 0.0f, 0.0f};
		// g_particles.color[index] = 0;
		// g_particles.age[index] = -1.0f;
		particles.velocity[index] = float3{0.0f, 0.0f, 0.0f};
	});


	g_lines.count = 0;

	RasterizationSettings rs_main;
	RasterizationSettings rs_left;
	RasterizationSettings rs_right;

	rs_main.texture   = nullptr;
	rs_main.colorMode = COLORMODE_VERTEXCOLOR;
	rs_main.world     = mat4::identity();
	rs_main.view      = uniforms.view;
	rs_main.proj      = uniforms.proj;
	rs_main.width     = uniforms.width;
	rs_main.height    = uniforms.height;

	if(uniforms.vrEnabled){
		rs_left = rs_main;
		rs_right = rs_main;

		rs_left.view    = uniforms.vr_left_view;
		rs_left.proj    = uniforms.vr_left_proj;
		rs_left.width   = uniforms.vr_left_width;
		rs_left.height  = uniforms.vr_left_height;

		rs_right.view   = uniforms.vr_right_view;
		rs_right.proj   = uniforms.vr_right_proj;
		rs_right.width  = uniforms.vr_right_width;
		rs_right.height = uniforms.vr_right_height;
	}
	
	grid.sync();

	{ // generate and draw a ground plane
		Triangles* triangles = createGroundPlane(50);
		Texture texture = createGridTexture();

		grid.sync();
		
		rs_main.colorMode = COLORMODE_TEXTURE;
		rs_main.texture = &texture;

		if(uniforms.vrEnabled){
			rs_left.colorMode  = COLORMODE_TEXTURE;
			rs_left.texture    = &texture;
			rs_right.colorMode = COLORMODE_TEXTURE;
			rs_right.texture   = &texture;

			rasterizeTriangles(triangles, fb_vr_left, rs_left);
			grid.sync();
			rasterizeTriangles(triangles, fb_vr_right, rs_right);
		}else{
			rasterizeTriangles(triangles, framebuffer, rs_main);
		}
	}

	grid.sync();

	{
		Triangles* sphere = createSphere(24);


		for(int i = 0; i <= 10; i++)
		for(int j = -10; j <= 10; j += 2)
		// for(int j : {0, 1, 2})
		// for(int i : {1})
		// for(int j : {-5})
		{
			// uint32_t X = hiprand(&thread_random_state) >> 16;
			// uint32_t Y = hiprand(&thread_random_state) >> 16;
			// uint32_t Z = hiprand(&thread_random_state) >> 16;
			// uint32_t upper = 1 << 16;

			// float x = float(X) / float(upper) - 0.5f;
			// float y = float(Y) / float(upper) - 0.5f;
			// float z = float(Z) / float(upper) - 0.5f;

			float x = i - 5;

			rs_main.texture = nullptr;
			rs_main.colorMode = COLORMODE_UV;
			rs_main.world = mat4::translate(x, -j - 1.0f, -0.3f) * mat4::scale(0.2f, 0.2f, 0.2f);
			

			if(uniforms.vrEnabled){
				RasterizationSettings rs_left;
				rs_left.texture   = nullptr;
				rs_left.colorMode = COLORMODE_UV;
				rs_left.world     = mat4::translate(x, -j - 1.0f, -0.3f) * mat4::scale(0.2f, 0.2f, 0.2f);
				rs_left.view      = uniforms.vr_left_view;
				rs_left.proj      = uniforms.vr_left_proj;
				rs_left.transform = uniforms.vr_left_transform;
				rs_left.width     = uniforms.vr_left_width;
				rs_left.height    = uniforms.vr_left_height;
				
				rasterizeTriangles(sphere, fb_vr_left, rs_left);
				grid.sync();

				RasterizationSettings rs_right;
				rs_right.texture   = nullptr;
				rs_right.colorMode = COLORMODE_VERTEXCOLOR;
				rs_right.world     = mat4::translate(x, -j - 1.0f, -0.3f) * mat4::scale(0.2f, 0.2f, 0.2f);
				rs_right.view      = uniforms.vr_right_view;
				rs_right.proj      = uniforms.vr_right_proj;
				rs_right.transform = uniforms.vr_right_transform;
				rs_right.width     = uniforms.vr_right_width;
				rs_right.height    = uniforms.vr_right_height;
				rasterizeTriangles(sphere, fb_vr_right, rs_right);

				// mat4 transform = uniforms.vr_right_transform;
				// // mat4 transform = uniforms.vr_right_proj * uniforms.vr_right_view;
				// float3 position = {x, -0.3f, -j - 1.0f};
				// rasterizeSprite(fb_vr_right, position, 0x000000ff, 10, transform, uniforms.vr_right_width, uniforms.vr_right_height);

				grid.sync();
			}else{
				// rasterizeTriangles(sphere, framebuffer, rs_main);
				float3 position = {x, -0.3f, -j - 1.0f};
				if(grid.thread_rank() == 0)
				rasterizeSprite(framebuffer, position, 0x000000ff, 10, uniforms.transform, uniforms.width, uniforms.height);
			}

			grid.sync();
		}
	}

	grid.sync();

	{ // DRAW CONTROLLERS
		Triangles* triangles    = allocator->alloc<Triangles*>(sizeof(Triangles));
		triangles->numTriangles = numTriangles;
		triangles->positions    = positions;
		triangles->uvs          = uvs;
		triangles->colors       = colors;

		Texture texture;
		texture.width  = 1024;
		texture.height = 1024;
		texture.data   = textureData;

		rs_main.texture    = &texture;
		rs_left.texture    = &texture;
		rs_right.texture   = &texture;
		
		rs_main.colorMode  = uniforms.colorMode;
		rs_left.colorMode  = uniforms.colorMode;
		rs_right.colorMode = uniforms.colorMode;

		{
			float s = 0.8f;
			mat4 rot = mat4::rotate(0.5f * PI, {1.0f, 0.0f, 0.0f}).transpose();
			mat4 translate = mat4::translate(0.0f, 0.0f, 0.0f);
			mat4 scale = mat4::scale(s, s, s);
			mat4 wiggle = mat4::rotate(cos(5.0f * uniforms.time) * 0.1f, {0.0f, 1.0f, 0.0f}).transpose();
			mat4 wiggle_yaw = mat4::rotate(cos(5.0f * uniforms.time) * 0.1f, {0.0f, 0.0f, 1.0f}).transpose();
			
			if(uniforms.vrEnabled){
				float sController = 0.05f;

				rs_left.world = rot * uniforms.vr_left_controller_pose.transpose() 
					* mat4::scale(sController, sController, sController);
				rs_right.world = rs_left.world;

				if(uniforms.vr_left_controller_active){
					rasterizeTriangles(triangles, fb_vr_left, rs_left);
					rasterizeTriangles(triangles, fb_vr_right, rs_right);
				}

				grid.sync();

				rs_left.world = rot * uniforms.vr_right_controller_pose.transpose() 
					* mat4::scale(sController, sController, sController);
				rs_right.world = rs_left.world;

				if(uniforms.vr_right_controller_active){
					rasterizeTriangles(triangles, fb_vr_left, rs_left);
					rasterizeTriangles(triangles, fb_vr_right, rs_right);
				}
			}else{
				rs_main.world = translate * wiggle * wiggle_yaw * rot * scale;
				rasterizeTriangles(triangles, framebuffer, rs_main);
			}

			grid.sync();
		}
	}

	grid.sync();

	// draw random points
	// for(int i = 0; i < 1; i++){
	// 	uint32_t X = hiprand(&thread_random_state) >> 16;
	// 	uint32_t Y = hiprand(&thread_random_state) >> 16;
	// 	uint32_t Z = hiprand(&thread_random_state) >> 16;
	// 	uint32_t upper = 1 << 16;

	// 	uint32_t color = hiprand(&thread_random_state);

	// 	color = SPECTRAL[color % 11];

	// 	float x = 10.0f * (float(X) / float(upper) - 0.5f);
	// 	float y =  1.0f * (float(Y) / float(upper) - 0.5f) - 0.8f;
	// 	float z = 10.0f * (float(Z) / float(upper) - 0.5f);

	// 	float3 position = {x, y, z};

	// 	rasterizePoint(fb_points, position, color, 
	// 		uniforms.transform, uniforms.width, uniforms.height);

	// 	// rasterizeSprite(fb_points, position, color, 5,
	// 	// 	uniforms.transform, uniforms.width, uniforms.height);
	// }

	// draw particles
	processRange(1'000'000, [&](int index){

		float age = particles.age[index];

		if(age == -1.0f) return;

		float3 position = particles.position[index];
		uint32_t color = particles.color[index];

		rasterizePoint(fb_points, position, color, 
			uniforms.transform, uniforms.width, uniforms.height);

	});

	grid.sync();

	// transfer points to main framebuffer
	processRange(uniforms.width * uniforms.height, [&](int pixelID){

		// atomicMin(&framebuffer[pixelID], fb_points[pixelID]);
		uint64_t closest = uint64_t(Infinity) << 32;

		int x = pixelID % int(uniforms.width);
		int y = pixelID / int(uniforms.width);

		int radius = 3;
		for(int ox = -radius; ox <= radius; ox++)
		for(int oy = -radius; oy <= radius; oy++)
		// for(int ox : {0})
		// for(int oy : {0})
		{
			int px = x + ox;
			int py = y + oy;

			if(px < 0 || px >= uniforms.width) continue;
			if(py < 0 || py >= uniforms.width) continue;

			int pid = px + int(uniforms.width) * py;

			uint64_t value = fb_points[pid];

			closest = min(closest, value);
		}

		atomicMin(&framebuffer[pixelID], closest);
		// fb_points[pixelID]

	});

	// if(grid.thread_rank() == 0){
	// 	drawBoundingBox({-2.0f, 0.0f, 0.0f}, {0.5f, 1.0f, 1.5f}, 0x000000ff);
	// }

	// grid.sync();

	// rasterizeLines(framebuffer, uniforms.transform, uniforms.width, uniforms.height);

}

extern "C" __global__
void kernel_draw_skybox(
	const Uniforms _uniforms,
	uint32_t* buffer,
	hipSurfaceObject_t gl_colorbuffer_main,
	hipSurfaceObject_t gl_colorbuffer_vr_left,
	hipSurfaceObject_t gl_colorbuffer_vr_right,
	uint64_t* framebuffer,
	uint64_t* fb_vr_left,
	uint64_t* fb_vr_right,
	uint32_t numTriangles,
	float3* positions,
	float2* uvs,
	uint32_t* colors,
	uint32_t* textureData,
	const Skybox skybox
){
	if(uniforms.vrEnabled){

		// TODO
		drawSkybox(
			uniforms.vr_left_proj, uniforms.vr_left_view, 
			uniforms.vr_left_proj_inv, uniforms.vr_left_view_inv, 
			fb_vr_left, 
			uniforms.vr_left_width, uniforms.vr_left_height, 
			skybox
		);

		drawSkybox(
			uniforms.vr_right_proj, uniforms.vr_right_view, 
			uniforms.vr_right_proj_inv, uniforms.vr_right_view_inv, 
			fb_vr_right, 
			uniforms.vr_right_width, uniforms.vr_right_height, 
			skybox
		);
		
		// if(grid.thread_rank() == 0){
		// 	mat4 mat = uniforms.vr_right_proj_inv;
		// 	printf("===========\n");
		// 	printf("%5.1f, %5.1f, %5.1f, %5.1f \n", mat[0].x, mat[0].y, mat[0].z, mat[0].w);
		// 	printf("%5.1f, %5.1f, %5.1f, %5.1f \n", mat[1].x, mat[1].y, mat[1].z, mat[1].w);
		// 	printf("%5.1f, %5.1f, %5.1f, %5.1f \n", mat[2].x, mat[2].y, mat[2].z, mat[2].w);
		// 	printf("%5.1f, %5.1f, %5.1f, %5.1f \n", mat[3].x, mat[3].y, mat[3].z, mat[3].w);
		// }

	}else{
		drawSkybox(
			uniforms.proj, uniforms.view, 
			uniforms.proj_inv, uniforms.view_inv, 
			framebuffer, 
			uniforms.width, uniforms.height, 
			skybox
		);
	}
}


extern "C" __global__
void kernel_toOpenGL(
	const Uniforms _uniforms,
	uint32_t* buffer,
	hipSurfaceObject_t gl_colorbuffer_main,
	hipSurfaceObject_t gl_colorbuffer_vr_left,
	hipSurfaceObject_t gl_colorbuffer_vr_right,
	uint64_t* framebuffer,
	uint64_t* fb_vr_left,
	uint64_t* fb_vr_right,
	uint32_t numTriangles,
	float3* positions,
	float2* uvs,
	uint32_t* colors,
	uint32_t* textureData,
	const Skybox skybox
){
	// TRANSFER TO OPENGL TEXTURE
	if(uniforms.vrEnabled){
		
		// left
		processRange(0, uniforms.vr_left_width * uniforms.vr_left_height, [&](int pixelIndex){
			int x = pixelIndex % int(uniforms.vr_left_width);
			int y = pixelIndex / int(uniforms.vr_left_width);

			uint64_t encoded = fb_vr_left[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;
			uint8_t* rgba = (uint8_t*)&color;
			uint32_t idepth = (encoded >> 32);
			float depth = *((float*)&idepth);

			if(EDL_ENABLED){
				float edlRadius = 2.0f;
				float edlStrength = 0.4f;
				float2 edlSamples[4] = {
					{-1.0f,  0.0f},
					{ 1.0f,  0.0f},
					{ 0.0f,  1.0f},
					{ 0.0f, -1.0f}
				};

				float sum = 0.0f;
				for(int i = 0; i < 4; i++){
					float2 samplePos = {
						x + edlSamples[i].x,
						y + edlSamples[i].y
					};

					int sx = clamp(samplePos.x, 0.0f, uniforms.vr_left_width - 1.0f);
					int sy = clamp(samplePos.y, 0.0f, uniforms.vr_left_height - 1.0f);
					int samplePixelIndex = sx + sy * uniforms.vr_left_width;

					uint64_t sampleEncoded = fb_vr_left[samplePixelIndex];
					uint32_t iSampledepth = (sampleEncoded >> 32);
					float sampleDepth = *((float*)&iSampledepth);

					sum += max(0.0, depth - sampleDepth);
				}

				float shade = exp(-sum * 300.0 * edlStrength);

				rgba[0] = float(rgba[0]) * shade;
				rgba[1] = float(rgba[1]) * shade;
				rgba[2] = float(rgba[2]) * shade;
			}

			// color = 0x000000ff;
			surf2Dwrite(color, gl_colorbuffer_vr_left, x * 4, y);
		});

		// right
		processRange(0, uniforms.vr_right_width * uniforms.vr_right_height, [&](int pixelIndex){
			int x = pixelIndex % int(uniforms.vr_right_width);
			int y = pixelIndex / int(uniforms.vr_right_width);

			uint64_t encoded = fb_vr_right[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;
			uint8_t* rgba = (uint8_t*)&color;
			uint32_t idepth = (encoded >> 32);
			float depth = *((float*)&idepth);

			if(EDL_ENABLED){
				float edlRadius = 2.0f;
				float edlStrength = 0.4f;
				float2 edlSamples[4] = {
					{-1.0f,  0.0f},
					{ 1.0f,  0.0f},
					{ 0.0f,  1.0f},
					{ 0.0f, -1.0f}
				};

				float sum = 0.0f;
				for(int i = 0; i < 4; i++){
					float2 samplePos = {
						x + edlSamples[i].x,
						y + edlSamples[i].y
					};

					int sx = clamp(samplePos.x, 0.0f, uniforms.vr_right_width - 1.0f);
					int sy = clamp(samplePos.y, 0.0f, uniforms.vr_right_height - 1.0f);
					int samplePixelIndex = sx + sy * uniforms.vr_right_width;

					uint64_t sampleEncoded = fb_vr_right[samplePixelIndex];
					uint32_t iSampledepth = (sampleEncoded >> 32);
					float sampleDepth = *((float*)&iSampledepth);

					sum += max(0.0, depth - sampleDepth);
				}

				float shade = exp(-sum * 300.0 * edlStrength);

				rgba[0] = float(rgba[0]) * shade;
				rgba[1] = float(rgba[1]) * shade;
				rgba[2] = float(rgba[2]) * shade;
			}

			// color = 0x000000ff;
			surf2Dwrite(color, gl_colorbuffer_vr_right, x * 4, y);
		});

		// blit vr displays to main window
		processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

			int x = pixelIndex % int(uniforms.width);
			int y = pixelIndex / int(uniforms.width);

			float u = fmodf(2.0 * float(x) / uniforms.width, 1.0f);
			float v = float(y) / uniforms.height;

			uint32_t color = 0x000000ff;
			if(x < uniforms.width / 2.0){
				int vr_x = u * uniforms.vr_left_width;
				int vr_y = v * uniforms.vr_left_height;
				int vr_pixelIndex = vr_x + vr_y * uniforms.vr_left_width;

				uint64_t encoded = fb_vr_left[vr_pixelIndex];
				color = encoded & 0xffffffffull;
			}else{
				int vr_x = u * uniforms.vr_right_width;
				int vr_y = v * uniforms.vr_right_height;
				int vr_pixelIndex = vr_x + vr_y * uniforms.vr_right_width;

				uint64_t encoded = fb_vr_right[vr_pixelIndex];
				color = encoded & 0xffffffffull;
			}

			// color = 0x000000ff;
			surf2Dwrite(color, gl_colorbuffer_main, x * 4, y);
		});

	}else{
		// blit custom cuda framebuffer to opengl texture
		processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex){

			int x = pixelIndex % int(uniforms.width);
			int y = pixelIndex / int(uniforms.width);

			uint64_t encoded = framebuffer[pixelIndex];
			uint32_t color = encoded & 0xffffffffull;

			surf2Dwrite(color, gl_colorbuffer_main, x * 4, y);
		});
	}
}